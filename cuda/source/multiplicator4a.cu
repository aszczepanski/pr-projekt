#include "hip/hip_runtime.h"
#include "multiplicator4a.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include "cuda_exceptions.h"
#include "array_manager.h"

template <int BLOCK_SIZE> __global__ void
MatrixMulKernel_4a(float *C, const float *A, const float *B, const int arraySize) {
    int bx = blockIdx.x;
    int by = blockIdx.y;

    int tx = threadIdx.x;
    int ty = threadIdx.y;

    int aBegin = arraySize * BLOCK_SIZE * by;
    int aEnd   = aBegin + arraySize - 1;
    int aStep  = BLOCK_SIZE;

    int bBegin = BLOCK_SIZE * bx;
    int bStep  = BLOCK_SIZE * arraySize;

    float Csub = 0.0f;
	
	float fetchA, fetchB;
	fetchA = A[aBegin + arraySize * ty + tx];
    fetchB = B[bBegin + arraySize * ty + tx];

    for (int a = aBegin, b = bBegin;
         a <= aEnd;
         a += aStep, b += bStep)
    {
        __shared__ float As[BLOCK_SIZE][BLOCK_SIZE];
        __shared__ float Bs[BLOCK_SIZE][BLOCK_SIZE];

		As[ty][tx] = fetchA;
		Bs[ty][tx] = fetchB;

        __syncthreads();

		fetchA = A[a + aStep + arraySize * ty + tx];
		fetchB = B[b + bStep + arraySize * ty + tx];

#pragma unroll
        for (int k = 0; k < BLOCK_SIZE; ++k) {
            Csub += As[ty][k] * Bs[k][tx];
        }

        __syncthreads();
    }

    int c = arraySize * BLOCK_SIZE * by + BLOCK_SIZE * bx;
    C[c + arraySize * ty + tx] = Csub;
}

void Multiplicator4a::launchKernel(const size_t arraySize, const size_t blockSize, ArrayManager* arrayManager) {
	clearErrorFlag();
	
	// Setup execution parameters
    dim3 threads(blockSize, blockSize);
    dim3 grid(ceil((float)arraySize/(float)blockSize), ceil((float)arraySize/(float)blockSize));

	// Execute the kernel
	if (blockSize == 8) {
        MatrixMulKernel_4a<8><<< grid, threads >>>(
			arrayManager->pointerToDev_C(), arrayManager->pointerToDev_A(), arrayManager->pointerToDev_B(), arraySize);
	} else if (blockSize == 16) {
        MatrixMulKernel_4a<16><<< grid, threads >>>(
			arrayManager->pointerToDev_C(), arrayManager->pointerToDev_A(), arrayManager->pointerToDev_B(), arraySize);
    } else if (blockSize == 22) {
        MatrixMulKernel_4a<22><<< grid, threads >>>(
			arrayManager->pointerToDev_C(), arrayManager->pointerToDev_A(), arrayManager->pointerToDev_B(), arraySize);
    }
	
    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "matrixMulCUDA launch failed: %s\n", hipGetErrorString(cudaStatus));
        throw CudaError(hipGetErrorString(hipGetLastError()));
    }
}